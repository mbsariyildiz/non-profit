#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>


bool allocate (void **ptr, size_t size){
    // since cudaMalloc accepts double pointer, casting is required.
    hipError_t stat = hipMalloc(ptr, size);

    if (stat == hipSuccess)
        return true;

    // if no success, print the error
    std::cout << "allocation stat: " <<  stat << std::endl;
    return false;
}

/*
enum cudaMemcpyKind:
cudaMemcpyHostToHost = 0
cudaMemcpyHostToDevice = 1
cudaMemcpyDeviceToHost = 2
cudaMemcpyDeviceToDevice = 3
cudaMemcpyDefault = 4
 */
bool copy(void *dst, void *src, size_t size, int cudaCpyKind){
    hipMemcpyKind dir;
    switch (cudaCpyKind) {
    case 0:
        dir = hipMemcpyHostToHost;
        break;
    case 1:
        dir = hipMemcpyHostToDevice;
        break;
    case 2:
        dir = hipMemcpyDeviceToHost;
        break;
    case 3:
        dir = hipMemcpyDeviceToDevice;
        break;
    default:
        dir = hipMemcpyHostToHost;
        break;
    }
    hipError_t stat = hipMemcpy(dst, src, size, dir);

    if (stat == hipSuccess)
        return true;

    // if no success, print the error
    std::cout << "copy stat: " <<  stat << std::endl;
    return false;
}

void release(void *ptr){ hipFree(ptr);}
